﻿/* Copyright (c) 2019, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

#if defined(WIN32) || defined(_WIN32) || defined(WIN64) || defined(_WIN64)
#define WINDOWS_LEAN_AND_MEAN
#define NOMINMAX
#include <windows.h>
#pragma warning(disable : 4819)
#endif

#include <Exceptions.h>
#include <ImageIO.h>
#include <ImagesCPU.h>
#include <ImagesNPP.h>

#include <string.h>
#include <fstream>
#include <iostream>
#include <vector>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <npp.h>        // umbrella include (pulls most of NPP)
#include <nppcore.h>    // for nppSetStream / nppGetStreamContext
#include <nppi.h>       // for image processing functions (resize, filter, etc.)

#include <hip/hip_runtime_api.h>
#include <helper_string.h>

#include "MipMapChunk.h"

/// <summary>
/// Creates a mipmap image from the source image using NPP's resize function.
/// </summary>
/// <param name="srcImage">The src image.</param>
/// <param name="streamCtx">The stream ctx.</param>
/// <param name="outDstImage">The pointer to the destination image.</param>
/// <param name="interpolationMode">The mode for interpolation.</param>
void createMipMap(
	npp::ImageNPP_8u_C4& srcImage, 
	NppStreamContext& streamCtx,
	npp::ImageNPP_8u_C4* outDstImage,
	int interpolationMode)
{
	if (srcImage.data() == NULL)
	{
		std::cout << "Failed to allocate device image memory." << std::endl;
		throw npp::Exception("Failed to allocate device image memory.");
	}


	NppiSize srcSize = { srcImage.width(), srcImage.height() };
	NppiRect srcROI = { 0, 0, srcSize.width, srcSize.height };

	// We start writing to right and top and then move Y for size of new mip map on each iteration.
	NppiSize finalSize = { srcSize.width * 1.5, srcSize.height };
	npp::ImageNPP_8u_C4 dstDeviceMemory(finalSize.width, finalSize.height);

	// Destination
	int step = 2;
	std::vector<MipMapChunk*> chunks;

	while (true)
	{
		int width = srcSize.width / step;
		int height = srcSize.height / step;

		// Create chunk and increase steps
		MipMapChunk* chunk = new MipMapChunk(srcImage, width, height, streamCtx, interpolationMode);
		step *= 2;
		chunks.push_back(chunk);

		// If 1x1 was last, break
		if (width <= 1 && height <= 1)
		{
			break;
		}
	}

	hipStreamSynchronize(streamCtx.hStream); // wait only for this stream

	// Copy source image first then chunks
	NPP_CHECK_NPP(nppiCopy_8u_C4R_Ctx(
		srcImage.data(), srcImage.pitch(),
		dstDeviceMemory.data(), dstDeviceMemory.pitch(),
		srcSize,
		streamCtx));

	// Copy chunks now to create mip map.
	int y = 0;
	for (auto& chunk : chunks)
	{
		// Copy chunk to final.
		NPP_CHECK_NPP(nppiCopy_8u_C4R_Ctx(
			chunk->GPUMemory.data(), chunk->GPUMemory.pitch(),
			dstDeviceMemory.data() + y * dstDeviceMemory.pitch() + srcSize.width * 4, dstDeviceMemory.pitch(),
			chunk->Size,
			streamCtx));

		y += chunk->Size.height;
	}

	*outDstImage = dstDeviceMemory;
}

bool initCudaAndSetupStream(NppStreamContext& nppStreamCtx)
{
	// Create a CUDA device and set it as the current device.
	int deviceCount = 0;
	NPP_CHECK_CUDA(hipGetDeviceCount(&deviceCount));

	if (deviceCount == 0)
	{
		std::cerr << "No CUDA devices found." << std::endl;
		return false;
	}

	NPP_CHECK_CUDA(hipGetDevice(&nppStreamCtx.nCudaDeviceId));

	nppStreamCtx.hStream = 0; // The NULL stream by default, set this to whatever your stream ID is if not the NULL stream.

	int driverVersion, runtimeVersion;
	hipDriverGetVersion(&driverVersion);
	hipRuntimeGetVersion(&runtimeVersion);

	printf("CUDA Driver  Version: %d.%d\n", driverVersion / 1000, (driverVersion % 100) / 10);
	printf("CUDA Runtime Version: %d.%d\n\n", runtimeVersion / 1000, (runtimeVersion % 100) / 10);

	NPP_CHECK_CUDA(hipDeviceGetAttribute(&nppStreamCtx.nCudaDevAttrComputeCapabilityMajor,
		hipDeviceAttributeComputeCapabilityMajor,
		nppStreamCtx.nCudaDeviceId));
	

	NPP_CHECK_CUDA(hipDeviceGetAttribute(&nppStreamCtx.nCudaDevAttrComputeCapabilityMinor,
		hipDeviceAttributeComputeCapabilityMinor,
		nppStreamCtx.nCudaDeviceId));

	NPP_CHECK_CUDA(hipStreamGetFlags(nppStreamCtx.hStream, &nppStreamCtx.nStreamFlags));

	hipDeviceProp_t oDeviceProperties;

	NPP_CHECK_CUDA(hipGetDeviceProperties(&oDeviceProperties, nppStreamCtx.nCudaDeviceId));

	nppStreamCtx.nMultiProcessorCount = oDeviceProperties.multiProcessorCount;
	nppStreamCtx.nMaxThreadsPerMultiProcessor = oDeviceProperties.maxThreadsPerMultiProcessor;
	nppStreamCtx.nMaxThreadsPerBlock = oDeviceProperties.maxThreadsPerBlock;
	nppStreamCtx.nSharedMemPerBlock = oDeviceProperties.sharedMemPerBlock;


	return true;
}


/// <summary>
/// Comma split string and fill in result array.
/// </summary>
/// <param name="str">The string to split by ','.</param>
/// <param name="result">The array where split parts fill be filled at.</param>
void commaSplitString(std::string str, std::vector<std::string>& result)
{
	std::string currentStr;

	for (int i = 0; i < str.length(); i++)
	{
		char c = str.c_str()[i];
		if (c == ',')
		{
			result.push_back(currentStr);
			currentStr = "";
		}
		else
		{
			currentStr += c;
		}
	}

	if (currentStr != "")
	{
		result.push_back(currentStr);
	}
}


bool getInputFileParameter(int argc, char* argv[], std::vector<std::string>& result)
{
	char* inputFilePath;

	if (checkCmdLineFlag(argc, (const char**)argv, "--input"))
	{
		getCmdLineArgumentString(argc, (const char**)argv, "--input", &inputFilePath);
	}
	else if (checkCmdLineFlag(argc, (const char**)argv, "--i"))
	{
		getCmdLineArgumentString(argc, (const char**)argv, "--i", &inputFilePath);
	}
	else
	{
		std::cout << "No input file specified. Use --input <filename> or --i <filename>." << std::endl;
		return false;
	}

	commaSplitString(std::string(inputFilePath), result);

	return true;
}

void getOutputFileParameter(int argc, char* argv[], std::vector<std::string>& result)
{
	char* outputFilePath = nullptr;

	if (checkCmdLineFlag(argc, (const char**)argv, "--output"))
	{
		getCmdLineArgumentString(argc, (const char**)argv, "--output", &outputFilePath);
	}
	else if (checkCmdLineFlag(argc, (const char**)argv, "--o"))
	{
		getCmdLineArgumentString(argc, (const char**)argv, "--o", &outputFilePath);
	}

	if (outputFilePath != nullptr)
	{
		commaSplitString(std::string(outputFilePath), result);
	}
}

void getInterpolationMode(int argc, char* argv[], int* outMode)
{
	int mode = NPPI_INTER_LINEAR;
	char* modeStr = nullptr;

	if (checkCmdLineFlag(argc, (const char**)argv, "--mode"))
	{
		getCmdLineArgumentString(argc, (const char**)argv, "--mode", &modeStr);
	}
	else if (checkCmdLineFlag(argc, (const char**)argv, "--m"))
	{
		getCmdLineArgumentString(argc, (const char**)argv, "--m", &modeStr);
	}

	if (modeStr == "1")
	{
		mode = NPPI_INTER_NN;
		std::cout << "MipMaps will be genereted with NearestNeighbour filter." << std::endl;
	}
	if (modeStr == "2")
	{
		mode = NPPI_INTER_CUBIC;
		std::cout << "MipMaps will be genereted with Cubic interpolation." << std::endl;

	}
	else
	{
		std::cout << "MipMaps will be genereted with Linear interpolation." << std::endl;
	}

	*outMode = mode;
}

std::string trimExtension(std::string str)
{
	int index = str.find_last_of('.');
	std::string subStr = str.substr(0, index);
	return subStr;
}


int main(int argc, char* argv[])
{
	printf("%s Starting...\n\n", argv[0]);

	NppStreamContext nppStreamCtx;
	if (!initCudaAndSetupStream(nppStreamCtx))
	{
		return EXIT_FAILURE;
	}

	// INPUT
	std::vector<std::string> inputs;
	if (!getInputFileParameter(argc, argv, inputs))
	{
		// Provide default argument if none are provided.
		std::cout << "Using ./data/Lena.png" << std::endl;
		inputs.push_back("./data/Lena.png");
	}

	// OUTPUT
	std::vector<std::string> outputs;
	getOutputFileParameter(argc, argv, outputs);

	// INTERPOLATION
	int mode;
	getInterpolationMode(argc, argv, &mode);

	try
	{
		for (int i = 0; i < inputs.size(); i++)
		{
			std::string input = inputs[i];

			// If there is output file name provided, use it instead.
			std::string output = trimExtension(input) + "_mipmap.png";
			if (outputs.size() > i)
			{
				output = outputs[i];
			}

			// Load image.
			npp::ImageCPU_8u_C4 inputHostMemory;
			if (npp::loadImage(input, &inputHostMemory))
			{
				std::cout << "Loaded image " << input << std::endl;
			}	
			else
			{
				std::cout << "Failed to load image " << input << std::endl;
				return EXIT_FAILURE;
			}

			// Generate mip map
			npp::ImageNPP_8u_C4 inputDeviceMemory(inputHostMemory);
			npp::ImageNPP_8u_C4 outputDeviceMemory;
			std::cout << "Creating mipmap for " << input << std::endl;
			createMipMap(inputDeviceMemory, nppStreamCtx, &outputDeviceMemory, mode);
			std::cout << "Mipmap created for " << input << std::endl;
			npp::ImageCPU_8u_C4 outputHostMemory(outputDeviceMemory.size());
			outputDeviceMemory.copyTo(outputHostMemory.data(), outputHostMemory.pitch());

			std::cout << "Saving MipMap" << output << std::endl;
			npp::saveImage(output, outputHostMemory);

		}
	}
	catch (npp::Exception& rException)
	{
		std::cerr << "Program error! The following exception occurred: \n";
		std::cerr << rException << std::endl;
		std::cerr << "Aborting." << std::endl;

		exit(EXIT_FAILURE);
	}
	catch (...)
	{
		std::cerr << "Program error! An unknown type of exception occurred. \n";
		std::cerr << "Aborting." << std::endl;

		exit(EXIT_FAILURE);
		return -1;
	}

	return 0;
}
